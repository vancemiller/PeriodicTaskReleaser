#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// CUDA utilities and system includes
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <sys/mman.h>
#include <hip/hip_runtime.h>
// includes, project
#include <helper_string.h>
#include <helper_functions.h> // includes for SDK helper functions
#include <hip/hip_runtime_api.h>    // includes for cuda initialization and error checking

#include "sf_kernels.h"

extern "C" {
#include "gpusync.h"
}

#define MAX_EPSILON_ERROR 5.0f
#define OFFSET(i) ((char *)NULL + (i))
#define MAX(a,b) ((a > b) ? a : b)
#define RADIUS 1
#ifdef FIXED_BLOCKWIDTH
#define BlockWidth 80
#define SharedPitch 384
#endif

const char *sSDKsample = "CUDA Sobel Edge-Detection";

static int imWidth  = 0;   // Image width
static int imHeight = 0;   // Image height
unsigned int g_Bpp;
unsigned int g_Index = 0;

unsigned char *pixels = NULL;  // Image pixel data on the host
float imageScale = 1.f;    // Image exposure

// Texture reference for reading image
texture<unsigned char, 2> tex;
extern __shared__ unsigned char LocalBlock[];
static hipArray *array = NULL;

// Stream for the thread's GPU Operations
hipStream_t stream;

// Device memory location for result
Pixel *d_result;
// Host memory location for result
unsigned char *h_result;

char dump_file[256];
char ref_image_path[] = "../Samples/Copy/SobelFilter/data/ref_shared.pgm";
// Sorry for this. Needs a path to the source image.
char image_path[] = "../Samples/Copy/SobelFilter/data/lena.pgm";

// Kernel execution parameters
dim3 threads;
#ifndef FIXED_BLOCKWIDTH
int BlockWidth;
#endif
dim3 blocks;
int SharedPitch;
int sharedMem;

// Utility functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
        file, line, (int)err, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

// override methods in hip/hip_runtime_api.h
template <class T> inline bool loadPGM(const char *file, T **data, 
  unsigned int *w, unsigned int *h) {
  unsigned char *idata = NULL;
  unsigned int channels;

  if (!__loadPPM(file, &idata, w, h, &channels)) {
    return false;
  }

  unsigned int size = *w **h * channels;

  // initialize mem if necessary
  // the correct size is checked / set in loadPGMc()
  if (NULL == *data) {
    checkCudaErrors(hipHostMalloc(data, sizeof(T) * size));
  }

  // copy and cast data
  std::transform(idata, idata + size, *data, ConverterFromUByte<T>());
  free(idata);
  return true;
}

template <class T> inline bool loadPPM4(const char *file, T **data,
  unsigned int *w,unsigned int *h) {
  unsigned char *idata = 0;
  unsigned int channels;

  if (__loadPPM(file, &idata, w, h, &channels)) {
    // pad 4th component
    int size = *w **h;
    // keep the original pointer
    unsigned char *idata_orig = idata;
    hipHostMalloc(data, sizeof(T) * size * 4);
    unsigned char *ptr = *data;

    for (int i=0; i<size; i++) {
      *ptr++ = *idata++;
      *ptr++ = *idata++;
      *ptr++ = *idata++;
      *ptr++ = 0;
    }

    free(idata_orig);
    return true;
  } else {
    free(idata);
    return false;
  }
}

// Kernels
__device__ unsigned char ComputeSobel(
    unsigned char ul, // upper left
    unsigned char um, // upper middle
    unsigned char ur, // upper right
    unsigned char ml, // middle left
    unsigned char mm, // middle (unused)
    unsigned char mr, // middle right
    unsigned char ll, // lower left
    unsigned char lm, // lower middle
    unsigned char lr, // lower right
    float fScale) {
  short Horz = ur + 2 * mr + lr - ul - 2 * ml - ll;
  short Vert = ul + 2 * um + ur - ll - 2 * lm - lr;
  short Sum = (short)(fScale * (abs((int)Horz) + abs((int)Vert)));

  if (Sum < 0) {
    return 0;
  } else if (Sum > 0xff) {
    return 0xff;
  }
  return (unsigned char) Sum;
}

__global__ void SobelShared(uchar4 *pSobelOriginal, unsigned short SobelPitch,
#ifndef FIXED_BLOCKWIDTH
    short BlockWidth, short SharedPitch,
#endif
    short w, short h, float fScale) {
  short u = 4 * blockIdx.x * BlockWidth;
  short v = blockIdx.y * blockDim.y + threadIdx.y;
  short ib;

  int SharedIdx = threadIdx.y * SharedPitch;

  for (ib = threadIdx.x; ib < BlockWidth + 2 * RADIUS; ib += blockDim.x) {
    LocalBlock[SharedIdx + 4 * ib + 0] = tex2D(tex,
       (float)(u + 4 * ib - RADIUS + 0), (float)(v - RADIUS));
    LocalBlock[SharedIdx + 4 * ib + 1] = tex2D(tex,
       (float)(u + 4 * ib - RADIUS + 1), (float)(v - RADIUS));
    LocalBlock[SharedIdx + 4 * ib + 2] = tex2D(tex,
       (float)(u + 4 * ib - RADIUS + 2), (float)(v - RADIUS));
    LocalBlock[SharedIdx + 4 * ib + 3] = tex2D(tex,
       (float)(u + 4 * ib - RADIUS + 3), (float)(v - RADIUS));
  }

  if (threadIdx.y < RADIUS * 2) {
    //
    // copy trailing RADIUS*2 rows of pixels into shared
    //
    SharedIdx = (blockDim.y + threadIdx.y) * SharedPitch;

    for (ib = threadIdx.x; ib < BlockWidth + 2 * RADIUS; ib += blockDim.x) {
      LocalBlock[SharedIdx + 4 * ib + 0] = tex2D(tex,
         (float)(u + 4 * ib - RADIUS + 0), (float)(v + blockDim.y - RADIUS));
      LocalBlock[SharedIdx + 4 * ib + 1] = tex2D(tex,
         (float)(u + 4 * ib - RADIUS + 1), (float)(v + blockDim.y - RADIUS));
      LocalBlock[SharedIdx + 4 * ib + 2] = tex2D(tex,
         (float)(u + 4 * ib - RADIUS + 2), (float)(v + blockDim.y - RADIUS));
      LocalBlock[SharedIdx + 4 * ib + 3] = tex2D(tex,
         (float)(u + 4 * ib - RADIUS + 3), (float)(v + blockDim.y - RADIUS));
    }
  }

  __syncthreads();

  u >>= 2;  // index as uchar4 from here
  uchar4 *pSobel = (uchar4 *)(((char *) pSobelOriginal) + v * SobelPitch);
  SharedIdx = threadIdx.y * SharedPitch;

  for (ib = threadIdx.x; ib < BlockWidth; ib += blockDim.x) {
    unsigned char pix00 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 0];
    unsigned char pix01 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 1];
    unsigned char pix02 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 2];
    unsigned char pix10 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 0];
    unsigned char pix11 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 1];
    unsigned char pix12 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 2];
    unsigned char pix20 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 0];
    unsigned char pix21 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 1];
    unsigned char pix22 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 2];

    uchar4 out;

    out.x = ComputeSobel(pix00, pix01, pix02,
       pix10, pix11, pix12,
       pix20, pix21, pix22, fScale);

    pix00 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 3];
    pix10 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 3];
    pix20 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 3];
    out.y = ComputeSobel(pix01, pix02, pix00,
       pix11, pix12, pix10,
       pix21, pix22, pix20, fScale);

    pix01 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 4];
    pix11 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 4];
    pix21 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 4];
    out.z = ComputeSobel(pix02, pix00, pix01,
       pix12, pix10, pix11,
       pix22, pix20, pix21, fScale);

    pix02 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 5];
    pix12 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 5];
    pix22 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 5];
    out.w = ComputeSobel(pix00, pix01, pix02,
       pix10, pix11, pix12,
       pix20, pix21, pix22, fScale);

    if (u + ib < w / 4 && v < h) {
      pSobel[u + ib] = out;
    }
  }

  __syncthreads();
}

__global__ void SobelCopyImage(Pixel *pSobelOriginal, unsigned int Pitch,
    int w, int h, float fscale) {
  unsigned char *pSobel = (unsigned char *) (((char *) pSobelOriginal) + blockIdx.x * Pitch);

  for (int i = threadIdx.x; i < w; i += blockDim.x) {
    pSobel[i] = min(max((tex2D(tex, (float) i, (float) blockIdx.x) * fscale), 0.f), 255.f);
  }
}

__global__ void SobelTex(Pixel *pSobelOriginal, unsigned int Pitch,
    int w, int h, float fScale) {
  unsigned char *pSobel = (unsigned char *)(((char *) pSobelOriginal) + blockIdx.x * Pitch);

  for (int i = threadIdx.x; i < w; i += blockDim.x) {
    unsigned char pix00 = tex2D(tex, (float) i - 1, (float) blockIdx.x - 1);
    unsigned char pix01 = tex2D(tex, (float) i + 0, (float) blockIdx.x - 1);
    unsigned char pix02 = tex2D(tex, (float) i + 1, (float) blockIdx.x - 1);
    unsigned char pix10 = tex2D(tex, (float) i - 1, (float) blockIdx.x + 0);
    unsigned char pix11 = tex2D(tex, (float) i + 0, (float) blockIdx.x + 0);
    unsigned char pix12 = tex2D(tex, (float) i + 1, (float) blockIdx.x + 0);
    unsigned char pix20 = tex2D(tex, (float) i - 1, (float) blockIdx.x + 1);
    unsigned char pix21 = tex2D(tex, (float) i + 0, (float) blockIdx.x + 1);
    unsigned char pix22 = tex2D(tex, (float) i + 1, (float) blockIdx.x + 1);
    pSobel[i] = ComputeSobel(pix00, pix01, pix02,
       pix10, pix11, pix12,
       pix20, pix21, pix22, fScale);
  }
}

// CPU code
void initializeData(char *file) {
  unsigned int w, h;
  size_t file_length = strlen(file);

  if (!strcmp(&file[file_length - 3], "pgm")) {
    if (loadPGM<unsigned char>(file, &pixels, &w, &h) != true) {
      printf("Failed to load PGM image file: %s\n", file);
      exit(EXIT_FAILURE);
    }

    g_Bpp = 1;
  }
  else if (!strcmp(&file[file_length - 3], "ppm")) {
    if (loadPPM4(file, &pixels, &w, &h) != true) {
      printf("Failed to load PPM image file: %s\n", file);
      exit(EXIT_FAILURE);
    }

    g_Bpp = 4;
  } else {
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();
    exit(EXIT_FAILURE);
  }

  imWidth = (int)w;
  imHeight = (int)h;
}

void loadDefaultImage() {
  initializeData(image_path);
}

void setupTexture(int iw, int ih, Pixel *data, int Bpp) {
  hipChannelFormatDesc desc;
  if (Bpp == 1) {
    desc = hipCreateChannelDesc<unsigned char>();
  } else {
    desc = hipCreateChannelDesc<uchar4>();
  }
  checkCudaErrors(hipMallocArray(&array, &desc, iw, ih));
}

// GPUSYNC interface

extern "C" void init(int sync_level) {
  /*
   * The sync_level parameter is an integer that indicates the desired level of
   * synchronization used by the GPU driver (values defined below).  The
   * specified level is used in hipSetDeviceFlags() to set the level
   * prior to initialization.
   */
  switch (sync_level) {
    case 0:
      hipSetDeviceFlags(hipDeviceScheduleSpin);
      break;
    case 1:
      hipSetDeviceFlags(hipDeviceScheduleYield);
      break;
    case 2:
      hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
      break;
    default:
      fprintf(stderr, "Unknown sync level: %d\n", sync_level);
      break;
  }
  // Follow convention and initialize CUDA/GPU
  // used here to invoke initialization of GPU locking
  hipFree(0);

  // Pin code
  if(!mlockall(MCL_CURRENT)) {
    fprintf(stderr, "Failed to lock code pages.\n");
    exit(EXIT_FAILURE);
  }
 
  // Set the device context 
  hipSetDevice(0);

  // create a user defined stream
  hipStreamCreate(&stream);

  sprintf(dump_file, "lena_shared.pgm");
}

extern "C" void mallocCPU(int num_elements) {
  loadDefaultImage(); 
  checkCudaErrors(hipHostMalloc(&h_result, imWidth * imHeight * sizeof(Pixel)));

  threads = dim3(16, 4);
#ifndef FIXED_BLOCKWIDTH
  BlockWidth = 80; // must be divisible by 16 for coalescing
#endif
  blocks = dim3(imWidth / (4 * BlockWidth) + (0 != imWidth % (4 * BlockWidth)),
      imHeight / threads.y + (0 != imHeight % threads.y));
  SharedPitch = ~0x3f & (4 * (BlockWidth + 2 * RADIUS) + 0x3f);
  sharedMem = SharedPitch * (threads.y + 2 * RADIUS);

  // for the shared kernel, width must be divisible by 4
  imWidth &= ~3;
}

extern "C" void mallocGPU(int num_elements) {
  setupTexture(imWidth, imHeight, pixels, g_Bpp);
  // may not be necessary
  // memset(pixels, 0x0, g_Bpp * sizeof(Pixel) * imWidth * imHeight);
  checkCudaErrors(hipMalloc((void **)&d_result, imWidth * imHeight * sizeof(Pixel)));
}

extern "C" void copyin(int num_elements) {
  checkCudaErrors(cudaMemcpyToArrayAsync(array, 0, 0, pixels,
      g_Bpp * sizeof(Pixel) * imWidth * imHeight, hipMemcpyHostToDevice, stream));
  checkCudaErrors(hipBindTextureToArray(tex, array));
  hipStreamSynchronize(stream);
}

extern "C" void exec(int num_elements) {
  SobelShared <<< blocks, threads, sharedMem, stream>>>((uchar4 *) d_result, imWidth,
#ifndef FIXED_BLOCKWIDTH
      BlockWidth, SharedPitch,
#endif
      imWidth, imHeight, imageScale);
  hipStreamSynchronize(stream);
}

extern "C" void copyout() {
  checkCudaErrors(hipMemcpyAsync(h_result, d_result, imWidth * imHeight * sizeof(Pixel), 
      hipMemcpyDeviceToHost, stream));
  hipStreamSynchronize(stream);
}

extern "C" void freeGPU() {
  checkCudaErrors(hipUnbindTexture(tex));
  checkCudaErrors(hipFree(d_result));
  checkCudaErrors(hipFreeArray(array));
}

extern "C" void freeCPU() {
  hipHostFree(h_result);
}

extern "C" void finish() {
  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  hipError_t err = hipDeviceReset();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deinitialize the device! (error code %s).\n", hipGetErrorString(err));
  }
  exit(EXIT_SUCCESS);
}

// int main(int argc, char **argv) {

//   printf("%s Starting...\n\n", sSDKsample);

//   printf("[%s] (automated testing w/ readback)\n", sSDKsample);
//   sf_init(0);
//   sf_mallocHost(argv[0]);
//   sf_cudaMalloc();
//   sf_copyin();
//   sf_exec();
//   sf_copyout();
//   printf("AutoTest %s done\n", argv[0]);
//   sdkSavePGM(dump_file, h_result, imWidth, imHeight);
//   sf_cudaFree();
//   sf_freeHost();
//   sf_finish(argv[0]);
// }

