#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/types.h>
#include <unistd.h>
#include <sched.h>
#include <errno.h>

extern "C" {
#include "../va.h"
}

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
    vectorAdd(const float *A, const float *B, float *C, int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements)
  {
    C[i] = A[i] + B[i];
  }
}

// Stream for the thread's GPU Operations
hipStream_t va_stream;

// Memory regions
float *h_vA, *h_vB, *h_vC;
float *d_vA, *d_vB, *d_vC;
size_t vector_len;
int v_threadsPerBlock;
int v_blocksPerGrid;

extern "C" void va_init(int sync_level) {
  /*
   * The sync_level parameter is an integer that indicates the desired level of
   * synchronization used by the GPU driver (values defined below).  The
   * specified level is used in hipSetDeviceFlags() to set the level
   * prior to initialization.
   */
  switch (sync_level) {
    case 0:
      hipSetDeviceFlags(hipDeviceScheduleSpin);
      break;
    case 1:
      hipSetDeviceFlags(hipDeviceScheduleYield);
      break;
    default:
      break;
  }

  // Follow convention and initialize CUDA/GPU
  // used here to invoke initialization of GPU locking
  hipFree(0);
  
  // create a user defined stream
  hipStreamCreate(&va_stream);
}

extern "C" void va_mallocHost(int numElements) {
  vector_len = numElements * sizeof(float);

  // Host allocations in pinned memory
  // Allocate the host input vector A
  hipError_t err = hipHostMalloc((void **) &h_vA, vector_len);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate host vector A (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Allocate the host input vector B
  err = hipHostMalloc((void **) &h_vB, vector_len);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate host vector B (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Allocate the host output vector C
  err = hipHostMalloc((void **)&h_vC, vector_len);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate host vector C (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) {
    h_vA[i] = rand()/(float)RAND_MAX;
    h_vB[i] = rand()/(float)RAND_MAX;
  }
  v_threadsPerBlock = 256;
  v_blocksPerGrid = (numElements + v_threadsPerBlock - 1) / v_threadsPerBlock;
}


extern "C" void va_cudaMalloc(int numElements) {
  // Allocate the device input vector A
  hipError_t err = hipMalloc((void **)&d_vA, vector_len);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Allocate the device input vector B
  err = hipMalloc((void **)&d_vB, vector_len);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Allocate the device output vector C
  err = hipMalloc((void **)&d_vC, vector_len);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
    return;
  }
}

extern "C" void va_copyin(int numElements) {
  // copy the A and B vectors from Host to Device memory
  // these calls are asynchronous so only the lock of CE can be handled in the wrapper
  hipError_t err = hipMemcpyAsync(d_vA, h_vA, vector_len, hipMemcpyHostToDevice, va_stream);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(va_stream);

  err = hipMemcpyAsync(d_vB, h_vB, vector_len, hipMemcpyHostToDevice, va_stream);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(va_stream);
}

extern "C" void va_exec(int numElements) {
  hipError_t err = hipSuccess;

  // Launch the Vector Add CUDA Kernel
  // lock of EE is handled in wrapper for hipLaunchByPtr()
  vectorAdd<<<v_blocksPerGrid, v_threadsPerBlock, 0, va_stream>>>(d_vA, d_vB, d_vC, numElements);

  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
    return;
  }
  // synchronize with the stream after kernel execution
  // the wrapper for this function releases any lock held (EE here)
  hipStreamSynchronize(va_stream);
}

extern "C" void va_copyout() {
  // Copy the result vector from Device to Host memory
  // This call is asynchronous so only the lock of CE can be handled in the wrapper
  hipError_t err = hipMemcpyAsync(h_vC, d_vC, vector_len, hipMemcpyDeviceToHost, va_stream);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(va_stream);
}

extern "C" void va_cudaFree() {
  // Free device global memory for inputs A and B and result C
  hipError_t err = hipFree(d_vA);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  err = hipFree(d_vB);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  err = hipFree(d_vC);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
    return;
  }
}

extern "C" void va_freeHost() {
  // Free host memory that was pinned
  hipHostFree(h_vA);
  hipHostFree(h_vB);
  hipHostFree(h_vC);
}
 
extern "C" void va_finish() {
  // clean up the user allocated stream
  hipStreamSynchronize(va_stream);
  hipStreamDestroy(va_stream);

  // Reset the device and return
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application returns
  hipError_t err = hipDeviceReset();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
  }
}
