/*
 * fastHog.cpp
 *
 *  Created on: May 14, 2009
 *      Author: viprad
 */
#include <stdio.h>
#include <stdlib.h>
#include "HOGEngine.h"
#include "HOGEngineDevice.h"
#include "HOGImage.h"
#include "Others/persondetectorwt.tcc"
extern "C" {
#include "../../gpusync.h"
}

HOGImage image;
hipStream_t stream;

char file_name[] = "Files/Images/testImage.bmp";

void init(int sync_level) {
  switch (sync_level) {
  case 0:
    hipSetDeviceFlags(hipDeviceScheduleSpin);
    break;
  case 1:
    hipSetDeviceFlags(hipDeviceScheduleYield);
    break;
  default:
    printf("Unknown sync level: %d\n", sync_level);
    break;
  }
  if (!HOGImageFile(file_name, &image)) {
    printf("Unable to load image file.\n");
    exit(1);
  }
  if (hipSetDevice(0) != hipSuccess) {
    printf("Unable to set cuda device.\n");
    exit(1);
  }
  if (hipFree(0) != hipSuccess) {
    printf("Error running hipFree(0).\n");
    exit(1);
  }
  if (hipStreamCreate(&stream) != hipSuccess) {
    printf("Unable to create cuda stream.\n");
    exit(1);
  }
  InitializeHOG(image.width, image.height, PERSON_LINEAR_BIAS,
    PERSON_WEIGHT_VEC, PERSON_WEIGHT_VEC_LENGTH);
}

void mallocCPU(int numElements) {
  HostAllocHOGEngineDeviceMemory();
}

void mallocGPU(int numElements) {
  DeviceAllocHOGEngineDeviceMemory();
}

void copyin(int numElements) {
  CopyInHOGEngineDevice();
}

void exec(int numElements) {
  // There are still memcpys to the device in HOGScale and HOGPadding--they
  // may require more work to get rid of because they seem to rely on variables
  // determined during the execution phase.
  BeginProcess(&image, -1, -1, -1, -1, -1.0f, -1.0f);
}

void copyout() {
  // TODO (Nathan): Split EndProcess into copyout() and finish(), remove disk
  // stuff.
  EndProcess();
}

void freeGPU() {
  DeviceFreeHOGEngineDeviceMemory();
}

void freeCPU() {
  HostFreeHOGEngineDeviceMemory();
}

void finish() {
  FinalizeHOG();
}

int main(void) {
  init(0);
  mallocCPU(0);
  mallocGPU(0);
  copyin(0);
  exec(0);
  copyout();
  freeGPU();
  freeCPU();
  finish();
}

/*
int main(void) {
  image = HOGImageFile(file_name);
  printf("Loaded Image\n");
  InitializeHOG(image->width, image->height, PERSON_LINEAR_BIAS,
    PERSON_WEIGHT_VEC, PERSON_WEIGHT_VEC_LENGTH);
  printf("Initialized HOG\n");
  BeginProcess(image, -1, -1, -1, -1, -1.0f, -1.0f);
  EndProcess();
  printf("Processed Image\n");
#ifdef FOOBAR
  FinalizeHOG();
#endif
  return 0;
}
*/
