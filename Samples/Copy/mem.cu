// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

extern "C" {
#include "../gpusync.h"
}

// Stream for the thread's GPU Operations
hipStream_t stream;

float *h;
float *d;
int size;

extern "C" void init(int sync_level) {
  /*
   * The sync_level parameter is an integer that indicates the desired level of
   * synchronization used by the GPU driver (values defined below).  The
   * specified level is used in hipSetDeviceFlags() to set the level
   * prior to initialization.
   */
  switch (sync_level) {
    case 0:
      hipSetDeviceFlags(hipDeviceScheduleSpin);
      break;
    case 1:
      hipSetDeviceFlags(hipDeviceScheduleYield);
      break;
    default:
      break;
  }
  // Follow convention and initialize CUDA/GPU
  // used here to invoke initialization of GPU locking
  hipFree(0);

  // create a user defined stream
  hipStreamCreate(&stream);
}

extern "C" void mallocCPU(int numElements) {
  hipError_t err = hipHostMalloc((void **) &h, numElements);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate host memory (error code %s)!\n", hipGetErrorString(err));
  }
}

extern "C" void mallocGPU(int numElements) {
  hipError_t err = hipSuccess;

  // Allocate device memory
  err = hipMalloc((void **) &d, numElements);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device memory (error code %s)!\n", hipGetErrorString(err));
    return;
  }
}

#define SPLITSIZE 8192
extern "C" void copyin(int numElements) {
  size = numElements;
  // these calls are asynchronous so only the lock of CE can be handled in the wrapper
  hipError_t err = hipMemcpyAsync(d, h, numElements, hipMemcpyHostToDevice, stream);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy memory from host to device (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(stream);
}

extern "C" void exec(int numElements) {
  // Nothing to do
}

extern "C" void copyout() {
  // this call is asynchronous so only the lock of CE can be handled in the wrapper
  hipError_t err = hipMemcpyAsync(h, d, size, hipMemcpyDeviceToHost, stream);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy memory from device to host (error code %s)!\n", hipGetErrorString(err));
    return;
  }
  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(stream);
}

extern "C" void freeGPU() {
  // Free device global memory for inputs A and B and result C
  hipError_t err = hipFree(d);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device memory A (error code %s)!\n", hipGetErrorString(err));
    return;
  }
}

extern "C" void freeCPU() {
  // Free host memory that was pinned
  hipHostFree(h);
}

extern "C" void finish() {
  // clean up the user allocated stream
  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  // Reset the device and return
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application returns
  hipError_t err = hipDeviceReset();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
  }
}
