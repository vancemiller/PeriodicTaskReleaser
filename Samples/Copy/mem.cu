// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

extern "C" {
#include "../mem.h"
}

// Stream for the thread's GPU Operations
hipStream_t mm_stream;

float *h_memtest;
float *d_memtest;

extern "C" void memtest_init(int sync_level, int numElements) {
  /*
   * The sync_level parameter is an integer that indicates the desired level of
   * synchronization used by the GPU driver (values defined below).  The
   * specified level is used in hipSetDeviceFlags() to set the level
   * prior to initialization.
   */
  switch (sync_level) {
    case 0:
      hipSetDeviceFlags(hipDeviceScheduleSpin);
      break;
    case 1:
      hipSetDeviceFlags(hipDeviceScheduleYield);
      break;
    default:
      break;
  }
  // Follow convention and initialize CUDA/GPU
  // used here to invoke initialization of GPU locking
  hipFree(0);

  // create a user defined stream
  hipStreamCreate(&mm_stream);

  hipError_t err = hipHostMalloc((void **) &h_memtest, numElements);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate host memory (error code %s)!\n", hipGetErrorString(err));
  }
}

extern "C" void memtest_alloc(int numElements) {
  hipError_t err = hipSuccess;

  // Allocate device memory
  err = hipMalloc((void **) &d_memtest, numElements);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device memory (error code %s)!\n", hipGetErrorString(err));
    return;
  }
}

#define SPLITSIZE 8192
extern "C" void memtest_copyin(int numElements) {
  // these calls are asynchronous so only the lock of CE can be handled in the wrapper
  hipError_t err = hipMemcpyAsync(d_memtest, h_memtest, numElements, hipMemcpyHostToDevice, mm_stream);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy memory from host to device (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(mm_stream);
}

extern "C" void memtest_copyout(int numElements) {
  // this call is asynchronous so only the lock of CE can be handled in the wrapper
  hipError_t err = hipMemcpyAsync(h_memtest, d_memtest, numElements, hipMemcpyDeviceToHost, mm_stream);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy memory from device to host (error code %s)!\n", hipGetErrorString(err));
    return;
  }
  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(mm_stream);
}

extern "C" void memtest_cudafree() {
  // Free device global memory for inputs A and B and result C
  hipError_t err = hipFree(d_memtest);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device memory A (error code %s)!\n", hipGetErrorString(err));
    return;
  }
}

extern "C" void memtest_cleanup() {
  // Free host memory that was pinned
  hipHostFree(h_memtest);

  // clean up the user allocated stream
  hipStreamSynchronize(mm_stream);
  hipStreamDestroy(mm_stream);

  // Reset the device and return
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application returns
  hipError_t err = hipDeviceReset();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
  }
}
